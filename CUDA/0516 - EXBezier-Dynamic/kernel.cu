#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define NUM_CRVS	1024		// 블록의 수 = 만들려는 베지에 곡선의 수 
#define	NUM_THREADS	32
#define MAX_SAMPLES 256

//병렬 Child를 쓸거면
//프로젝트 속성 Cuda C++ Common에서 Generate ~를 예로 한다.

// 2차 베지에 곡선을 표현하는 구조체
struct BzrCrv {
	float2 CP[3];	// 제어점 3개 배열
	int nVertices;	// 곡선 위의 샘플링 점의 수 (4 ~ 256)
	float2* vertexPos;	// 곡선 위의 샘플링된 점의 좌표.. 동적 할당한다.
};


__global__ void computerBzrCrvs_Parent(BzrCrv* Crvs, int NumCrvs);
__global__ void computeBzrCrvs_Child(int _i, BzrCrv* Crvs, int _numSamples);
__global__ void freeVertexMem(BzrCrv* Crv, int numCrvs);


__device__ float2 operator +(float2 a, float2 b)
{
	float2 c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}
__device__ float2 operator -(float2 a, float2 b)
{
	float2 c;
	c.x = a.x - b.x;
	c.y = a.y - b.y;
	return c;
}

__device__ float2 operator *(float a, float2 b)
{
	float2 c;
	c.x = a * b.x;
	c.y = a * b.y;
	return c;
}

__device__ float length(float2 a)
{
	float c = sqrtf(a.x * a.x + a.y * a.y);
	return c;
}


int main()
{
	BzrCrv* Crvs = new BzrCrv[NUM_CRVS];

	//베지에 커브의 CP[3]에 임이의 점을 넣어준다.
	float2 last = { 0, 0 };
	for (int i = 0; i < NUM_CRVS; ++i)
	{
		Crvs[i].CP[0] = last;
		for (int j = 1; j < 3; ++j)
		{
			Crvs[i].CP[j].x = (float)rand() / (float)RAND_MAX; // 0.0 ~ 1.0 사이의 임의 x 좌표
			Crvs[i].CP[j].y = (float)rand() / (float)RAND_MAX; // 0.0 ~ 1.0 사이의 임의 y 좌표
		}
		last = Crvs[i].CP[2]; //n번째 베지에 곡선의 끝점 = n+1번째 베지에 곡선의 시작점
	}


	BzrCrv* Dev_Crvs;	// 전역 메모리 주소
	hipSetDevice(0);
	hipMalloc((void**)&Dev_Crvs, sizeof(BzrCrv) * NUM_CRVS);
	hipMemcpy(Dev_Crvs, Crvs, sizeof(BzrCrv) * NUM_CRVS, hipMemcpyHostToDevice);
	
	// 커널 함수 호출......
	clock_t st = clock();
	computerBzrCrvs_Parent << <NUM_CRVS / NUM_THREADS, NUM_THREADS >> > (Dev_Crvs, NUM_CRVS); //이전 것은 블록 하나가 곡선 하나를 담당했는데 지금 것은 쓰레드하나가 곡선 하나를 담당
	hipDeviceSynchronize();
	hipMemcpy(Crvs, Dev_Crvs, sizeof(BzrCrv) * NUM_CRVS, hipMemcpyDeviceToHost); //이부분이 문제 왜냐하면 BzrCrv의 float2*는 동적할당 포인터이기 때문에
	clock_t ed = clock();
	printf("Elapsed time = %u ms. \n", ed - st);
	hipDeviceReset();

	for (int i = 0; i < Crvs[0].nVertices; ++i)
	{
		printf("(x, y) = (%f, %f) \n", Crvs[0].vertexPos[i].x, Crvs[0].vertexPos[i].y);
	}

	// 동적 할당 해제
	freeVertexMem << <NUM_CRVS / NUM_THREADS, NUM_THREADS >> > (Crvs,NUM_CRVS);
	hipFree(Dev_Crvs);
	delete[] Crvs;
	return 0;
}


//쓰레드 하나가 곡선 하나를 맡는다.
__global__ void computerBzrCrvs_Parent(BzrCrv* Crvs, int NumCrvs)
{
	int i = blockIdx.x * NUM_THREADS + threadIdx.x;

	Crvs[i].nVertices = MAX_SAMPLES; // 각 곡선에서 256개로 샘플링. (곡률에 따라 달라지면 좋지만 그건 힘드니...)
	hipMalloc((void**)&Crvs[i].vertexPos, Crvs[i].nVertices * sizeof(float2)); //VertexPos 동적 할당
	computeBzrCrvs_Child << <ceil((float)Crvs[i].nVertices/NUM_THREADS),NUM_THREADS >> > (i, Crvs, Crvs[i].nVertices);


	//for (int j = 0; j < MAX_SAMPLES; j += NUM_THREADS)
	//{
	//	int idx = j + threadIdx.x;  // j = 0 -> idx = 0 ~ 31, j = 32 -> idx = 32 ~ 64, 
	//	if (idx < MAX_SAMPLES)
	//	{
	//		float t = (float)idx / (float)(MAX_SAMPLES - 1);  // 0.0 < t < 1.0
	//		float c0 = (1.0f - t) * (1.0f - t);
	//		float c1 = 2.0 * t * (1.0f - t);
	//		float c2 = t * t;
	//		float2 pos = c0 * Crvs[i].CP[0] + c1 * Crvs[i].CP[1] + c2 * Crvs[i].CP[2];
	//		Crvs[i].vertexPos[idx] = pos;
	//	}
	//}
}

__global__ void computeBzrCrvs_Child(int _i, BzrCrv* Crvs, int _numSamples)
{
		int idx = NUM_THREADS * blockIdx.x + threadIdx.x;  // j = 0 -> idx = 0 ~ 31, j = 32 -> idx = 32 ~ 64, 
		float t = (float)idx / (float)(MAX_SAMPLES - 1);  // 0.0 < t < 1.0
		float c0 = (1.0f - t) * (1.0f - t);
		float c1 = 2.0 * t * (1.0f - t);
		float c2 = t * t;
		float2 pos = c0 * Crvs[_i].CP[0] + c1 * Crvs[_i].CP[1] + c2 * Crvs[_i].CP[2];
		Crvs[_i].vertexPos[idx] = pos;
}

__global__ void freeVertexMem(BzrCrv* Crvs, int numCrvs)
{
	int i = NUM_THREADS * blockIdx.x + threadIdx.x;
	hipFree(Crvs[i].vertexPos);
}