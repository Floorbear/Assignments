﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>

#define IDX2C(i,j,ld)	((j)*(ld) +(i))

void main()
{
	//CPU 메모리 할당
	int M = 2, K = 3, N = 2;
	float* A = new float[M * K];
	float* B = new float[K * N];
	float* C = new float[M * N];

	//A행렬 초기화
	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < K; ++j)
			A[IDX2C(i, j, M)] = rand() % 10;
	}

	//B행렬 초기화
	for (int i = 0; i < K; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			B[IDX2C(i, j, K)] = rand() % 10;
		}
	}

	//GPU 메모리 할당
	float* dev_A = nullptr;
	float* dev_B = nullptr;
	float* dev_C = nullptr;

	hipMalloc((void**)&dev_A, M * K * sizeof(float));
	hipMalloc((void**)&dev_B, K * N * sizeof(float));
	hipMalloc((void**)&dev_C, M * N * sizeof(float));
	hipMemset(dev_C, 0, M * N * sizeof(float));

	//cublas 핸들 생성
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//CPU 데이터를 GPU로 복사
	hipblasSetMatrix(M, K, sizeof(float), A, M, dev_A, M);
	hipblasSetMatrix(K, N, sizeof(float), B, K, dev_B, K);

	//3단계 cublas 함수 호출 (alpha * A * B + beta * C)
	const float alpha = 1.0, beta = 0.0;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		M, N, K,
		&alpha, dev_A, M,
		dev_B, K,
		&beta, dev_C, M);

	// GPU -> CPU로 결과 복사 
	hipblasGetMatrix(M, N, sizeof(float), dev_C, M, C, M);

	//핸들 제거
	hipblasDestroy(handle);

	//행렬 A 출력
	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < K; ++j)
		{
			printf("%5.0f ", A[IDX2C(i, j, M)]);
		}
		printf("\n");
	}

	printf("\n");

	//행렬 B 출력
	for (int i = 0; i <K; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%5.0f ", B[IDX2C(i, j, K)]);
		}
		printf("\n");
	}

	printf("\n");

	//행렬 C 출력
	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%5.0f ", C[IDX2C(i, j, M)]);
		}
		printf("\n");
	}

	delete[] A;
	delete[] B;
	delete[] C;
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

}

