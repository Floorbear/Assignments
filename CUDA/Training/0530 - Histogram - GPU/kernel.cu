﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 1024 * 1024 *1024
#define NUMTHREAD 256

__global__ void Histo_kernel(unsigned int* _histogram, unsigned char* _buffer, int _size)
{
	__shared__ unsigned int tmp_histo[256];
	tmp_histo[threadIdx.x] = 0;
	__syncthreads();

	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int i = bx * blockDim.x + tx;

	atomicAdd(&tmp_histo[_buffer[i]], 1);
	//_histogram[_buffer[i]]++;
	__syncthreads();

	atomicAdd(&_histogram[tx], tmp_histo[tx]);

}

int main()
{
	//0 ~ 255 값으로 초기화된 버퍼 배열 생성
	unsigned char* pBuffer = new unsigned char[SIZE];
	for (int i = 0; i < SIZE; ++i)
	{
		pBuffer[i] = rand() % 256;
	}
	unsigned int Histogram[256] = { 0, };


	//사용할 GPU 선택하고, 메모리 할당
	hipSetDevice(0);
	unsigned char* dev_buffer = nullptr;
	hipMalloc((void**)&dev_buffer, sizeof(unsigned char) * SIZE);
	hipMemcpy(dev_buffer, pBuffer, sizeof(unsigned char) * SIZE, hipMemcpyHostToDevice);

	unsigned int* dev_histogram = nullptr;
	hipMalloc((void**)&dev_histogram, sizeof(unsigned int) * 256);
	hipMemset(dev_histogram, 0, sizeof(unsigned int) * 256); // 값을 0으로 초기화

	printf("Start.... \n");
	clock_t st = clock();
	Histo_kernel<<<SIZE / NUMTHREAD,NUMTHREAD>>>(dev_histogram, dev_buffer, SIZE);
	hipDeviceSynchronize();
	clock_t ed = clock();
	hipMemcpy(Histogram, dev_histogram, sizeof(unsigned int) * 256, hipMemcpyDeviceToHost);
	printf("Elapsed Time : %u ms\n", ed - st);

	//결과 검증
	int sum = 0;
	for (int i = 0; i < 256; ++i)
	{
		sum = sum + Histogram[i];
	}
	printf("Sum = %d \n", sum);
	delete[] pBuffer;
	hipFree(dev_buffer);
	hipFree(dev_histogram);
}