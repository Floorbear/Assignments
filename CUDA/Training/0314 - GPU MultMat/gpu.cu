#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>
#define INDEX 128

//void Init(float* _P, int _Width, int _Value)
//{
//	for (int i = 0; i < _Width; i++)
//	{
//		_P[i] = rand() % 3 - 1.0;
//	}
//}
void MultMatCPU(float* _P, float* M, float* _N, int _Width)
{
	for (int i = 0; i < _Width; i++)
	{
		for (int j = 0; j < _Width; j++)
		{
			float Sum = 0.0f;
			for (int k = 0; k < _Width; k++)
			{
				//2차원 배열을 1차원 배열로 인덱싱 하는 것 :: !! 이해 잘해 !!
				Sum = Sum + M[i * _Width + k] * _N[k * _Width + j];
			}
			_P[i * _Width + j] = Sum;
		}
	}
}

__global__ void MultMatGPU(float* _P, float* M, float* _N, int _Width) //여기 포인터는 GPU쪽 주소여야 한다.
{
	//!!!!!이미지 좌표처럼 X가 가로, Y가 세로가 된다. i가 행이니 Y, j가 열이니 X가 된다. 헷갈리니 주의하자.!!!!!!!!
	int i = blockIdx.y * 32 +threadIdx.y;
	int j = blockIdx.x * 32 +threadIdx.x;
	float Sum = 0;
	for (int k = 0; k < _Width; k++)
	{
		//2차원 배열을 1차원 배열로 인덱싱 하는 것 :: !! 이해 잘해 !!
		Sum = Sum + M[i * _Width + k] * _N[k * _Width + j];
	}
	_P[i * _Width + j] = Sum;
	
}

void PrintMat(float* _P, int _Width)
{
	std::cout << "------------------------------------------------------------------------------------------" << std::endl;
	for (int i = 0; i < _Width; i++)
	{
		for (int j = 0; j < _Width; j++)
		{
			printf("%.1f       ", _P[i * _Width + j]);
		}
		std::cout << std::endl;
	}
}
int main()
{
	float* arSum = new float[INDEX * INDEX];
	float* arM = new float[INDEX * INDEX];
	float* arN = new float[INDEX * INDEX];

	for (int i = 0; i < INDEX * INDEX; i++)
	{
		arM[i] = rand() % 3 - 1.0;
		arN[i] = rand() % 3 - 1.0;
		arSum[i] = 0;
	}

	hipSetDevice(0);

	float* devM;
	float* devN;
	float* devP;

	//1. 동적 할당
	hipMalloc((void**)&devM, INDEX * INDEX * sizeof(float));
	hipMalloc((void**)&devN, INDEX * INDEX * sizeof(float));
	hipMalloc((void**)&devP, INDEX * INDEX * sizeof(float));

	//2. CPU 데이터를 GPU에 복사
	hipMemcpy(devM, arM, INDEX * INDEX * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devN, arN, INDEX * INDEX * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devP, arSum, INDEX * INDEX * sizeof(float), hipMemcpyHostToDevice);

	//3. 커널 함수 수행
	dim3 gridDim(INDEX / 32, INDEX / 32, 1);
	dim3 blockDim(32, 32);
	std::chrono::system_clock::time_point St = std::chrono::system_clock::now();
	MultMatGPU << <gridDim,blockDim >> > (devP, devM, devN, INDEX);
	//MultMatCPU(arSum, arM, arN, INDEX);
	hipError_t Status = hipDeviceSynchronize();
	if (Status != hipSuccess)
	{
		printf("Error");
	}
	hipMemcpy(arSum, devP, INDEX * INDEX * sizeof(float), hipMemcpyDeviceToHost);

	std::chrono::system_clock::time_point Ed = std::chrono::system_clock::now();

	std::chrono::nanoseconds Time = Ed - St;
	std::chrono::microseconds microTime = std::chrono::duration_cast<std::chrono::microseconds>((Ed - St));
	std::chrono::milliseconds milliTime = std::chrono::duration_cast<std::chrono::milliseconds>((Ed - St));
	PrintMat(arSum, INDEX);

	//printf("Elapsed Time = %lld nanoseconds \n \n", Time.count());
	printf("Elapsed Time = %lld microseconds \n \n", microTime.count());
	//printf("Elapsed Time = %lld milliseconds \n \n", milliTime.count());



	delete[] arSum;
	delete[] arM;
	delete[] arN;
	hipFree(devM);
	hipFree(devN);
	hipFree(devP);

}

