#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>
#define WIDTH 1001
#define TILE_WIDTH 32

//void Init(float* _P, int _Width, int _Value)
//{
//	for (int i = 0; i < _Width; i++)
//	{
//		_P[i] = rand() % 3 - 1.0;
//	}
//}
void MultMatCPU(float* _P, float* M, float* _N, int _Width)
{
	for (int i = 0; i < _Width; i++)
	{
		for (int j = 0; j < _Width; j++)
		{
			float Sum = 0.0f;
			for (int k = 0; k < _Width; k++)
			{
				//2차원 배열을 1차원 배열로 인덱싱 하는 것 :: !! 이해 잘해 !!
				Sum = Sum + M[i * _Width + k] * _N[k * _Width + j];
			}
			_P[i * _Width + j] = Sum;
		}
	}
}

__global__ void MultMatGPU(float* _P, float* M, float* _N, int _Width) //여기 포인터는 GPU쪽 주소여야 한다.
{
	//!!!!!이미지 좌표처럼 X가 가로, Y가 세로가 된다. i가 행이니 Y, j가 열이니 X가 된다. 헷갈리니 주의하자.!!!!!!!!
	int i = blockIdx.y * TILE_WIDTH +threadIdx.y;
	int j = blockIdx.x * TILE_WIDTH +threadIdx.x;
	float Sum = 0;
	if (i < WIDTH && j < WIDTH) //WIDTH가 TILE_WIDTH의 배수가 아닐 때, 추가로 생성된 블록이 이상한 주소를 참조하는것을 방지하기 위해!!
	{
		for (int k = 0; k < _Width; k++)
		{
			//2차원 배열을 1차원 배열로 인덱싱 하는 것 :: !! 이해 잘해 !! && 내적 파트임
			Sum = Sum + M[i * _Width + k] * _N[k * _Width + j];
		}
		_P[i * _Width + j] = Sum;

	}
}

void PrintMat(float* _P, int _Width)
{
	std::cout << "------------------------------------------------------------------------------------------" << std::endl;
	for (int i = 0; i < _Width; i++)
	{
		for (int j = 0; j < _Width; j++)
		{
			printf("%.1f       ", _P[i * _Width + j]);
		}
		std::cout << std::endl;
	}
}
int main()
{
	float* arSum = new float[WIDTH * WIDTH];
	float* arM = new float[WIDTH * WIDTH];
	float* arN = new float[WIDTH * WIDTH];

	for (int i = 0; i < WIDTH * WIDTH; i++)
	{
		arM[i] = rand() % 3 - 1.0;
		arN[i] = rand() % 3 - 1.0;
		arSum[i] = 0;
	}

	hipSetDevice(0);

	float* devM;
	float* devN;
	float* devP;

	//1. 동적 할당
	hipMalloc((void**)&devM, WIDTH * WIDTH * sizeof(float));
	hipMalloc((void**)&devN, WIDTH * WIDTH * sizeof(float));
	hipMalloc((void**)&devP, WIDTH * WIDTH * sizeof(float));

	//2. CPU 데이터를 GPU에 복사
	hipMemcpy(devM, arM, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devN, arN, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devP, arSum, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);

	//3. 커널 함수 수행
	dim3 gridDim((WIDTH - 1) / TILE_WIDTH + 1, (WIDTH - 1 ) / TILE_WIDTH + 1, 1);
	dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
	std::chrono::system_clock::time_point St = std::chrono::system_clock::now();
	MultMatGPU << <gridDim,blockDim >> > (devP, devM, devN, WIDTH);
	//MultMatCPU(arSum, arM, arN, INDEX);
	hipError_t Status = hipDeviceSynchronize();
	if (Status != hipSuccess)
	{
		printf("Error");
	}
	hipMemcpy(arSum, devP, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost);

	std::chrono::system_clock::time_point Ed = std::chrono::system_clock::now();

	std::chrono::nanoseconds Time = Ed - St;
	std::chrono::microseconds microTime = std::chrono::duration_cast<std::chrono::microseconds>((Ed - St));
	std::chrono::milliseconds milliTime = std::chrono::duration_cast<std::chrono::milliseconds>((Ed - St));
	PrintMat(arSum, WIDTH);

	//printf("Elapsed Time = %lld nanoseconds \n \n", Time.count());
	printf("Elapsed Time = %lld microseconds \n \n", microTime.count());
	//printf("Elapsed Time = %lld milliseconds \n \n", milliTime.count());



	delete[] arSum;
	delete[] arM;
	delete[] arN;
	hipFree(devM);
	hipFree(devN);
	hipFree(devP);

}

