#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>
#define WIDTH 1024
#define TILE_WIDTH 32

//void Init(float* _P, int _Width, int _Value)
//{
//	for (int i = 0; i < _Width; i++)
//	{
//		_P[i] = rand() % 3 - 1.0;
//	}
//}
void MultMatCPU(float* _P, float* M, float* _N, int _Width)
{
	for (int i = 0; i < _Width; i++)
	{
		for (int j = 0; j < _Width; j++)
		{
			float Sum = 0.0f;
			for (int k = 0; k < _Width; k++)
			{
				//2차원 배열을 1차원 배열로 인덱싱 하는 것 :: !! 이해 잘해 !!
				Sum = Sum + M[i * _Width + k] * _N[k * _Width + j];
			}
			_P[i * _Width + j] = Sum;
		}
	}
}

__global__ void MultMatGPU(float* _P, float* M, float* _N, int _Width) //여기 포인터는 GPU쪽 주소여야 한다.
{
	//!!!!!이미지 좌표처럼 X가 가로, Y가 세로가 된다. i가 행이니 Y, j가 열이니 X가 된다. 헷갈리니 주의하자.!!!!!!!!
	int i = blockIdx.y * TILE_WIDTH +threadIdx.y;
	int j = blockIdx.x * TILE_WIDTH +threadIdx.x;
	float Sum = 0;
	if (i < WIDTH && j < WIDTH) //WIDTH가 TILE_WIDTH의 배수가 아닐 때, 추가로 생성된 블록이 이상한 주소를 참조하는것을 방지하기 위해!!
	{
		for (int k = 0; k < _Width; k++)
		{
			//2차원 배열을 1차원 배열로 인덱싱 하는 것 :: !! 이해 잘해 !! && 내적 파트임
			Sum = Sum + M[i * _Width + k] * _N[k * _Width + j];
		}
		_P[i * _Width + j] = Sum;

	}
}

__global__ void MultMatGPU_Shared(float* P, float* M, float* N, int _width)
{
	__shared__ float _M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float _N[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;
	int i = TILE_WIDTH * by + ty;
	int j = TILE_WIDTH * bx + tx;

	float sum = 0.f;
	for (int m = 0; m < _width / TILE_WIDTH; ++m)
	{
		_M[ty][tx] = M[i * _width + m * TILE_WIDTH + tx];
		_N[ty][tx] = N[(m * TILE_WIDTH + ty)*_width + j];
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k)
		{
			sum = sum + _M[ty][k] * _N[k][tx];
		}
		__syncthreads();
	}
	P[i * _width + j] = sum;
}

void PrintMat(float* _P, int _Width)
{
	std::cout << "------------------------------------------------------------------------------------------" << std::endl;
	for (int i = 0; i < _Width; i++)
	{
		for (int j = 0; j < _Width; j++)
		{
			printf("%.1f       ", _P[i * _Width + j]);
		}
		std::cout << std::endl;
	}
}
int main()
{
	float* arSum = new float[WIDTH * WIDTH];
	float* arM = new float[WIDTH * WIDTH];
	float* arN = new float[WIDTH * WIDTH];

	for (int i = 0; i < WIDTH * WIDTH; i++)
	{
		arM[i] = rand() % 3 - 1.0;
		arN[i] = rand() % 3 - 1.0;
		arSum[i] = 0;
	}

	hipSetDevice(0);

	float* devM;
	float* devN;
	float* devP;

	//1. 동적 할당
	hipMalloc((void**)&devM, WIDTH * WIDTH * sizeof(float));
	hipMalloc((void**)&devN, WIDTH * WIDTH * sizeof(float));
	hipMalloc((void**)&devP, WIDTH * WIDTH * sizeof(float));

	//2. CPU 데이터를 GPU에 복사
	hipMemcpy(devM, arM, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devN, arN, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devP, arSum, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);

	//3. 전역 메모리 행렬 곱
	{
		dim3 gridDim((WIDTH - 1) / TILE_WIDTH + 1, (WIDTH - 1) / TILE_WIDTH + 1, 1);
		dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
		std::chrono::system_clock::time_point St = std::chrono::system_clock::now();
		MultMatGPU << <gridDim, blockDim >> > (devP, devM, devN, WIDTH);
		//MultMatCPU(arSum, arM, arN, INDEX);
		hipError_t Status = hipDeviceSynchronize();
		if (Status != hipSuccess)
		{
			printf("Error");
		}
		hipMemcpy(arSum, devP, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost);

		std::chrono::system_clock::time_point Ed = std::chrono::system_clock::now();

		std::chrono::nanoseconds Time = Ed - St;
		std::chrono::microseconds microTime = std::chrono::duration_cast<std::chrono::microseconds>((Ed - St));
		std::chrono::milliseconds milliTime = std::chrono::duration_cast<std::chrono::milliseconds>((Ed - St));
		//PrintMat(arSum, WIDTH);

		//printf("Elapsed Time = %lld nanoseconds \n \n", Time.count());
		printf("GPU :: Elapsed Time = %lld microseconds \n \n", microTime.count());
		//printf("Elapsed Time = %lld milliseconds \n \n", milliTime.count());
	}

	//4. 공유 메모리 행렬 곱 릴리즈 모드시 더 빠름
	{
		dim3 gridDim((WIDTH - 1) / TILE_WIDTH + 1, (WIDTH - 1) / TILE_WIDTH + 1, 1);
		dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
		std::chrono::system_clock::time_point St = std::chrono::system_clock::now();
		MultMatGPU_Shared << <gridDim, blockDim >> > (devP, devM, devN, WIDTH);
		//MultMatCPU(arSum, arM, arN, INDEX);
		hipError_t Status = hipDeviceSynchronize();
		if (Status != hipSuccess)
		{
			printf("Error");
		}
		hipMemcpy(arSum, devP, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost);

		std::chrono::system_clock::time_point Ed = std::chrono::system_clock::now();

		std::chrono::nanoseconds Time = Ed - St;
		std::chrono::microseconds microTime = std::chrono::duration_cast<std::chrono::microseconds>((Ed - St));
		std::chrono::milliseconds milliTime = std::chrono::duration_cast<std::chrono::milliseconds>((Ed - St));
		//PrintMat(arSum, WIDTH);

		//printf("Elapsed Time = %lld nanoseconds \n \n", Time.count());
		printf("GPU Shared :: Elapsed Time = %lld microseconds \n \n", microTime.count());
		//printf("Elapsed Time = %lld milliseconds \n \n", milliTime.count());
	}





	delete[] arSum;
	delete[] arM;
	delete[] arN;
	hipFree(devM);
	hipFree(devN);
	hipFree(devP);

}

