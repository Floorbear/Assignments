#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <crt/hip/device_functions.h>
#define Size 1024 * 1024 * 1024
#define NumThreads 512

__global__ void parallel_sum1(float* dev_M, int size);//바로 인접한 인덱스를 더하는 것
__global__ void parallel_sum2(float* dev_M, int size);//멀리있는 녀석 들고와서 더하는 것

int main()
{
	float* M = new float[Size];
	for (int i = 0; i < Size; ++i)
	{
		M[i] = rand() / (float)RAND_MAX;
	}

	hipSetDevice(0);
	float* dev_M1;
	float* dev_M2;

	hipMalloc((void**)&dev_M1, sizeof(float) * Size);
	hipMalloc((void**)&dev_M2, sizeof(float) * Size);
	hipMemcpy(dev_M1, M, sizeof(float) * Size, hipMemcpyHostToDevice);
	hipMemcpy(dev_M2, M, sizeof(float) * Size, hipMemcpyHostToDevice);
	printf("초기화 완료...\n\n");

	//CPU 순차합
	clock_t st = clock();
	double sum = 0.0;
	for (int i = 0; i < Size; ++i)
	{
		sum += M[i];
	}
	clock_t ed = clock();
	printf("cpu 경과 시간 = %u ms \n", ed - st);
	printf("순차합 = %lf\n", sum);

	//쓰레드 구성 정의
	dim3 grid(Size / NumThreads, 1, 1);
	dim3 block(NumThreads, 1, 1);

	//병렬합 (분기)
	st = clock();
	parallel_sum1 << <grid, block >> > (dev_M1, Size);
	hipDeviceSynchronize();
	hipMemcpy(M, dev_M1, grid.x * sizeof(float), hipMemcpyDeviceToHost);
	sum = 0.0;
	for (int i = 0; i < grid.x; ++i)
	{
		sum += M[i];
	}
	ed = clock();
	printf("GPU 경과 시간 = %u ms\n", ed - st);
	printf("병렬합(분기) = %lf\n",sum);

	//병렬합 (덜)
	st = clock();
	parallel_sum2 << <grid, block >> > (dev_M2, Size);
	hipDeviceSynchronize();
	hipMemcpy(M, dev_M2, grid.x * sizeof(float), hipMemcpyDeviceToHost);
	sum = 0.0;
	for (int i = 0; i < grid.x; ++i)
	{
		sum += M[i];
	}
	ed = clock();
	printf("GPU 경과 시간 = %u ms\n", ed - st);
	printf("병렬합(덜분기) = %lf\n", sum);

	//메모리 해제 
	delete[] M;
	hipFree(dev_M1);
	hipFree(dev_M2);
}


__global__ void parallel_sum1(float* dev_M, int size)
{
	__shared__ float partialSum[NumThreads];//공유 메모리의 단위는 블럭! 
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;
	partialSum[tid] = dev_M[i];
	__syncthreads();

	//병렬 reduction 계산
	for (int s = 1; s < blockDim.x; s *= 2)
	{
		if ((tid % (2 * s)) == 0)
		{
			partialSum[tid] += partialSum[tid + s];
		}
		__syncthreads();
	}


	if (tid == 0)
	{
		dev_M[blockIdx.x] = partialSum[0];
	}
}
__global__ void parallel_sum2(float* dev_M, int size)
{
	__shared__ float partialSum[NumThreads];//공유 메모리의 단위는 블럭! 
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;
	partialSum[tid] = dev_M[i];
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s /= 2)
	{
		if (tid < s)
		{
			partialSum[tid] += partialSum[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		dev_M[blockIdx.x] = partialSum[0];
	}
}