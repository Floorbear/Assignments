﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

# define INDEXCOUNT 1024

void AddCPU(float* _Des, float* _Target1, float* _Target2, int _IndexCount)
{
    for (int i = 0; i < _IndexCount; i++)
    {
        _Des[i] = _Target1[i] + _Target2[i];
    }
}

__global__ void AddGPU(float* _Des, float* _Target1, float* _Target2)
{
    int Block = blockIdx.x;
    int i = Block * 1024 + threadIdx.x;
    _Des[i] = _Target1[i] + _Target2[i];
}

void PrintArray(float* _Array, int _IndexCount)
{
    for (int i = 0; i < _IndexCount; i++)
    {
        printf("%.5f   ", _Array[i]);
    }
    printf("\n");
}

int main()
{
    hipError_t Status = hipSetDevice(0);

    assert(Status == hipSuccess);

    float* Arr1 = new float[INDEXCOUNT];
    float* Arr2 = new float[INDEXCOUNT];
    float* ArrSum = new float[INDEXCOUNT];

    for (int i = 0; i < INDEXCOUNT; i++)
    {
        Arr1[i] = rand() / static_cast<float>(RAND_MAX);
        Arr2[i] = -Arr1[i];
        ArrSum[i] = -1.f;
    }


    float* GPU_Arr1;
    float* GPU_Arr2;
    float* GPU_ArrSum;

    //메모리 할당
    Status = hipMalloc((void**)&GPU_Arr1, INDEXCOUNT * sizeof(float));
    Status = hipMalloc((void**)&GPU_Arr2, INDEXCOUNT * sizeof(float));
    Status = hipMalloc((void**)&GPU_ArrSum, INDEXCOUNT * sizeof(float));

    //GPU에 메모리 복사
    Status = hipMemcpy(GPU_Arr1, Arr1, INDEXCOUNT * sizeof(float), hipMemcpyHostToDevice);
    Status = hipMemcpy(GPU_Arr2, Arr2, INDEXCOUNT * sizeof(float), hipMemcpyHostToDevice);
    Status = hipMemcpy(GPU_ArrSum, ArrSum, INDEXCOUNT * sizeof(float), hipMemcpyHostToDevice);


    //PrintArray(Arr1, INDEXCOUNT);
    //PrintArray(Arr2, INDEXCOUNT);
    clock_t cStart = clock();
    //AddCPU(ArrSum, Arr1, Arr2, INDEXCOUNT);
    //!! 중요 !!
    //! 한 블록 당 쓰레드의 최대 개수가 존재한다 (대략 1024개 이상)
    //! 따라서 쓰레드 수가 많아질 것같으면 블록을 추가해야 한다.
    //! 몇번째 블록인지  blockIdx를 통해 확인 가능하다
    AddGPU<<<INDEXCOUNT / 1024,1024>>>(GPU_ArrSum, GPU_Arr1, GPU_Arr2);
    Status = hipDeviceSynchronize();
    clock_t cEnd = clock();


    Status = hipMemcpy(ArrSum, GPU_ArrSum, INDEXCOUNT * sizeof(float), hipMemcpyDeviceToHost);
    PrintArray(ArrSum, INDEXCOUNT);
    // PrintArray(ArrSum, INDEXCOUNT);

    printf("경과 시간 : %u (ms)", cEnd - cStart);

    delete[] Arr1;
    delete[] Arr2;
    delete[] ArrSum;
    hipFree(GPU_Arr1);
    hipFree(GPU_Arr2);
    hipFree(GPU_ArrSum);

    return 0;
}

