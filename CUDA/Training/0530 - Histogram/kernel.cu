#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define SIZE 1024 * 1024 * 1024

void main()
{

	//0 ~ 255 값으로 초기화된 버퍼 배열 생성
	unsigned char* pBuffer = new unsigned char[SIZE];
	for (int i = 0; i < SIZE; ++i)
	{
		pBuffer[i] = rand() % 256;
	}

	// 히스토그램 계산
	unsigned int Histogram[256] = { 0, };
	printf("Start.... \n");
	clock_t st = clock();
	for (int i = 0; i < SIZE; ++i)
	{
		Histogram[pBuffer[i]]++;
	}
	clock_t ed = clock();
	printf("Elaspsed Time = %u ms \n", ed - st);

	int sum = 0;
	for (int i = 0; i < 256; ++i)
	{
		sum = sum + Histogram[i];
	}
	printf("Sum = %d \n", sum);
	delete[] pBuffer;
}