#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define NUM_CRVS	1024 * 1024		// 블록의 수 = 만들려는 베지에 곡선의 수 
#define	NUM_THREADS	32
#define MAX_SAMPLES 256

// 2차 베지에 곡선을 표현하는 구조체
struct BzrCrv {
	float2 CP[3];	// 제어점 3개 배열
	int nVertices;	// 곡선 위의 샘플링 점의 수 (4 ~ 256)
	float2 vertexPos[MAX_SAMPLES];	// 곡선 위의 샘플링된 점의 좌표
};

//하나의 베지에 곡선은 3개의 정점을 필요로 한다.
//NUM_CRVS 만큼의 베지에 곡선을 동적생성한다.
//생성된 베지에 곡선마다 정점을 넣어준다.
//생성된 베지에 곡선을 매게변수로 커널함수를 돌린다.
// 
//한 블록은 하나의 베지에 곡선을 정의한다.
//한 블록당 베지에 샘플링(베지에 곡선의 정점 수)은 256개다.
//한 블록당 쓰레드는 32개다
//쓰레드 하나당 베지에 정점 8개를 정의 해야한다.

__global__ void computerBzrCrvs(BzrCrv* Crvs, int NumCrvs);

__device__ float2 operator +(float2 a, float2 b)
{
	float2 c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}
__device__ float2 operator -(float2 a, float2 b)
{
	float2 c;
	c.x = a.x - b.x;
	c.y = a.y - b.y;
	return c;
}

__device__ float2 operator *(float a, float2 b)
{
	float2 c;
	c.x = a * b.x;
	c.y = a * b.y;
	return c;
}

__device__ float length(float2 a)
{
	float c = sqrtf(a.x * a.x + a.y * a.y);
	return c;
}


int main()
{
	BzrCrv* Crvs = new BzrCrv[NUM_CRVS];

	//베지에 커브의 CP[3]에 임이의 점을 넣어준다.
	float2 last = { 0, 0 };
	for (int i = 0; i < NUM_CRVS; ++i)
	{
		Crvs[i].CP[0] = last;
		for (int j = 1; j < 3; ++j)
		{
			Crvs[i].CP[j].x = (float)rand() / (float)RAND_MAX; // 0.0 ~ 1.0 사이의 임의 x 좌표
			Crvs[i].CP[j].y = (float)rand() / (float)RAND_MAX; // 0.0 ~ 1.0 사이의 임의 y 좌표
		}
		last = Crvs[i].CP[2]; //n번째 베지에 곡선의 끝점 = n+1번째 베지에 곡선의 시작점
	}

	BzrCrv* Dev_Crvs;	// 전역 메모리 주소
	hipSetDevice(0);
	hipMalloc((void**)&Dev_Crvs, sizeof(BzrCrv) * NUM_CRVS);
	hipMemcpy(Dev_Crvs, Crvs, sizeof(BzrCrv) * NUM_CRVS, hipMemcpyHostToDevice);

	// 커널 함수 호출......
	clock_t st = clock();
	computerBzrCrvs << <NUM_CRVS, NUM_THREADS >> > (Dev_Crvs, NUM_CRVS);
	hipMemcpy(Crvs, Dev_Crvs, sizeof(BzrCrv) * NUM_CRVS, hipMemcpyDeviceToHost);
	clock_t ed = clock();
	printf("Elapsed time = %u ms. \n", ed - st);
	hipDeviceReset();

	for (int i = 0; i < Crvs[0].nVertices; ++i)
	{
		printf("(x, y) = (%f, %f) \n", Crvs[0].vertexPos[i].x, Crvs[0].vertexPos[i].y);
	}

	// 동적 할당 해제
	hipFree(Dev_Crvs);
	delete[] Crvs;
	return 0;
}


//각 블록은 하나의 베지에 곡선을 정의한다.
//한 블록당 베지에 샘플링은 256개다.
//한 블록당 쓰레드는 32개다
//쓰레드 하나당 베지에 정점 8개를 정의 해야한다.
__global__ void computerBzrCrvs(BzrCrv* Crvs, int NumCrvs)
{
	int i = blockIdx.x;
	if (i < NumCrvs)
	{
		Crvs[i].nVertices = MAX_SAMPLES; // 각 곡선에서 256개로 샘플링.

		for (int j = 0; j < MAX_SAMPLES; j += NUM_THREADS)
		{
			int idx = j + threadIdx.x;  // j = 0 -> idx = 0 ~ 31, j = 32 -> idx = 32 ~ 64, 
			if (idx < MAX_SAMPLES)
			{
				float t = (float)idx / (float)(MAX_SAMPLES - 1);  // 0.0 < t < 1.0
				float c0 = (1.0f - t) * (1.0f - t);
				float c1 = 2.0 * t * (1.0f - t);
				float c2 = t * t;
				float2 pos = c0 * Crvs[i].CP[0] + c1 * Crvs[i].CP[1] + c2 * Crvs[i].CP[2];
				Crvs[i].vertexPos[idx] = pos;
			}
		}
	}
}

